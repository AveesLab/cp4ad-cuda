#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>


__global__ void conv_kernel( uint8_t* dev_input, uint8_t* dev_output, int8_t* FILTER, int width_in, int height_in, int width_out, int height_out, int in_channel, int out_channel, int padding, int stride, int kernel_size ) {
    int gy = blockIdx.y * blockDim.y + threadIdx.y; 
    int gx = blockIdx.x * blockDim.x + threadIdx.x;

    if (gx < width_out && gy < height_out) {
        int sum = 0;
        int cx = stride * gx + (kernel_size / 2);
        int cy = stride * gy + (kernel_size / 2);
        for (int dy = -(kernel_size / 2); dy <= (kernel_size / 2); dy++) {
            for (int dx = -(kernel_size / 2); dx <= (kernel_size / 2); dx++) {
                sum += (dev_input[width_in * (cy + dy) + (cx + dx)] * FILTER[kernel_size * ((kernel_size / 2) + dy) + (kernel_size / 2) + dx]);
            }
        }
        if (sum < 0) sum = 0;
        else if (sum > 255) sum = 255;
        int out_idx = gx + gy * width_out;
        dev_output[out_idx] = sum;
    }
}

void conv_gpu(uint8_t* input, uint8_t* output, int width_in, int height_in, int width_out, int height_out, int in_channel, int out_channel, int padding, int stride, int kernel_size) {
	int8_t Filter[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};

    uint8_t* dev_input = NULL;
    uint8_t* dev_output = NULL;
	int8_t* dev_Filter = NULL;
  
    hipMalloc( (void**)&dev_input, width_in * height_in * in_channel * sizeof(uint8_t) );
    hipMalloc( (void**)&dev_output, width_out * height_out * out_channel * sizeof(uint8_t) );
    
	hipMalloc( (void**)&dev_Filter, 9 * sizeof(int8_t) );
    hipMemcpy( dev_input, input, width_in * height_in * in_channel * sizeof(uint8_t), hipMemcpyHostToDevice );
	
	hipMemcpy( dev_Filter, Filter, 9 * sizeof(int8_t), hipMemcpyHostToDevice );

    dim3 dimBlock(32, 32, 1);
    dim3 dimGrid((width_out + dimBlock.x - 1)/ dimBlock.x, (height_out + dimBlock.y - 1) / dimBlock.y, 1);
	
	clock_t start = clock();
    conv_kernel<<<dimGrid, dimBlock>>>(dev_input, dev_output, dev_Filter, width_in, height_in, width_out, height_out, in_channel, out_channel, padding, stride, kernel_size);
	hipDeviceSynchronize();
	clock_t end = clock();

    hipMemcpy( output, dev_output, width_out * height_out * out_channel * sizeof(uint8_t), hipMemcpyDeviceToHost );
	
    hipFree( dev_input );
	hipFree( dev_Filter );
	hipFree( dev_output );
	
	double execution_time = (double) (end - start) / CLOCKS_PER_SEC;
	printf("Execution time: %d usec\n", (int) (execution_time * 1000000));
}
