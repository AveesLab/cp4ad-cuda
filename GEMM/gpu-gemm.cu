#include "hip/hip_runtime.h"
#include "utils.c"


int alpha = 1;
int beta = 1;
int matsize = 100; 


__global__ void kernelGEMM( float* C, float* A, float* B, int matsize, int alpha, int beta ) {
	int gy = blockDim.y * blockIdx.y + threadIdx.y;
	int gx = blockDim.x * blockIdx.x + threadIdx.x;
	if (gy < matsize && gx < matsize) {
		float sum = 0.0f;
		for (int k = 0; k < matsize; k++) {
			int idxA = gy * matsize + k;
			int idxB = k * matsize + gx;
			sum += A[idxA] * B[idxB];
		}
		int idxC = gy * matsize + gx;
		C[idxC] = alpha * sum + beta * C[idxC];
	}
}

int main(void) {
    float matA[matsize * matsize];
    float matB[matsize * matsize];
    float matC[matsize * matsize]; // = {0, 0, 0, 0, .....}
	// printMat( "Before_matC", matC, matsize, matsize );
	srand( 0 );
	setNormalizedRandomData( matA, matsize * matsize );
	setNormalizedRandomData( matB, matsize * matsize );
	float* dev_matA = NULL;
	float* dev_matB = NULL;
	float* dev_matC = NULL;
	// Write your code below. Hint1. Memory allocation to GPU
	hipMalloc( (void**)&dev_matA, matsize * matsize * sizeof(float) );
	hipMalloc( (void**)&dev_matB, matsize * matsize * sizeof(float) );
	hipMalloc( (void**)&dev_matC, matsize * matsize * sizeof(float) );
	
	// Write your code below. Hint2 Memory Copy CPU to GPU
	hipMemcpy( dev_matA, matA, matsize * matsize * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_matB, matB, matsize * matsize * sizeof(float), hipMemcpyHostToDevice );

	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid((matsize + dimBlock.x - 1) / dimBlock.x, (matsize + dimBlock.y - 1) / dimBlock.y, 1);
	clock_t start = clock();
	kernelGEMM <<< dimGrid, dimBlock>>>(dev_matC, dev_matA, dev_matB, matsize, alpha, beta );
	hipDeviceSynchronize();
	clock_t end = clock();
	// Write your code below. Hint3. Memory Copy GPU to CPU
	hipMemcpy( matC, dev_matC, matsize * matsize * sizeof(float), hipMemcpyDeviceToHost );	
	// Write your code below. Hint4. Delete GPU's Memory
	hipFree( dev_matA );
	hipFree( dev_matB );
	hipFree( dev_matC );
	double execution_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Execution time: %d usec\n", (int) (execution_time * 1000000));
	printf("matrix size = matsize * matsize = %d * %d\n", matsize, matsize);
	printMat( "matA", matA, matsize, matsize );
	printMat( "matB", matB, matsize, matsize );
	printMat( "matC", matC, matsize, matsize );
	return 0;
}
