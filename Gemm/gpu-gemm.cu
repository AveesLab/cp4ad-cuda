#include "hip/hip_runtime.h"
#include "utils.c"


int alpha = 1;
int beta = 1;
int matsize = 100; 


__global__ void kernelGEMM( float* C, float* A, float* B, int matsize, int alpha, int beta ) {
	// Write below code. //
	int gy = blockIdx.y * blockDim.y + threadIdx.y;
	int gx = blockIdx.x * blockDim.x + threadIdx.x;
	if (gy < matsize && gx < matsize) {
		float sum = 0.0f;
		for (int k = 0; k < matsize; ++k) {
			int idxA = gy * matsize + k;
			int idxB = k * matsize + gx;
			sum += A[idxA] * B[idxB];
		}
		int idxC = gy * matsize + gx;
		C[idxC] = alpha * sum + beta * C[idxC];
	}
}

int main(void) {
    float matA[matsize * matsize];
    float matB[matsize * matsize];
    float matC[matsize * matsize];
	// printMat( "Before_matC", matC, matsize, matsize );
	srand( 0 );
	setNormalizedRandomData( matA, matsize * matsize );
	setNormalizedRandomData( matB, matsize * matsize );
	float* dev_matA = NULL;
	float* dev_matB = NULL;
	float* dev_matC = NULL;
	//Memory allocation to GPU
	hipMalloc( (void**)&dev_matA, matsize * matsize * sizeof(float) );
	hipMalloc( (void**)&dev_matB, matsize * matsize * sizeof(float) );
	hipMalloc( (void**)&dev_matB, matsize * matsize * sizeof(float) );
	//Memory Copy CPU to GPU
	hipMemcpy( dev_matA, matA, matsize * matsize * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_matB, matB, matsize * matsize * sizeof(float), hipMemcpyHostToDevice );
	hipMemcpy( dev_matC, matC, matsize * matsize * sizeof(float), hipMemcpyHostToDevice );


	dim3 dimBlock(32, 32, 1);
	dim3 dimGrid((matsize + dimBlock.x - 1) / dimBlock.x, (matsize + dimBlock.y - 1) / dimBlock.y, 1);
	clock_t start = clock();
	kernelGEMM <<< dimGrid, dimBlock>>>(dev_matC, dev_matA, dev_matB, matsize, alpha, beta );
	hipDeviceSynchronize();
	clock_t end = clock();
	//Memory Copy GPU to CPU
	hipMemcpy( matC, dev_matC, matsize * matsize * sizeof(float), hipMemcpyDeviceToHost );
	//Delete GPU's Memory
	hipFree( dev_matA );
	hipFree( dev_matB );
	hipFree( dev_matC );

	double execution_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Execution time: %d usec\n", (int) (execution_time * 1000000));
	printf("matrix size = matsize * matsize = %d * %d\n", matsize, matsize);
	printMat( "matA", matA, matsize, matsize );
	printMat( "matB", matB, matsize, matsize );
	printMat( "matC", matC, matsize, matsize );
	return 0;
}
