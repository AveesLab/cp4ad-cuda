
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 256

__global__ void ReverseKernel(int *arr)
{
	__shared__ int temp[N];
	int idx = threadIdx.x;
	int idx_inv = N - idx - 1;
	temp[idx] = arr[idx];

	//__syncthreads();

	arr[idx] = temp[idx_inv];
}

int main(void)
{
	int h_arr[N], result_arr[N], comp_arr[N];

	for (int i = 0; i < N; i++)
	{
		h_arr[i] = i;
		comp_arr[i] = N - i - 1;
	}

	int *d_arr;
	hipMalloc( (void**)&d_arr, N * sizeof(int) );

	hipMemcpy( d_arr, h_arr, N * sizeof(int), hipMemcpyHostToDevice );
	ReverseKernel<<<1, N>>>(d_arr);
	//cudaDeviceSynchronize();
	hipMemcpy( result_arr, d_arr, N * sizeof(int), hipMemcpyDeviceToHost );
	for (int i = 0; i < N; i++)
	{
		if (result_arr[i] != comp_arr[i]) 
		{
			printf("result_arr[%d] : %d\n", i, result_arr[i]);
			printf("comp_arr[%d] : %d\n", i, comp_arr[i]);
			printf("result_arr[%d] != comp_arr[%d]\n", i, i);
			printf("\n");
		}
	}
	
	hipFree(d_arr);

	return 0;
}
