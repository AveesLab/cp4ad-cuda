
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void SyncKernel(void) {
	printf("THREAD %d in Block (%d, %d)\n", threadIdx.x * blockDim.y + threadIdx.y, blockIdx.x, blockIdx.y);
}

int main(void)
{
	dim3 dimBlock(2, 2);
	dim3 dimGrid(2, 2);

	printf("CUDA kernel launch with (%d * %d) blocks of (%d * %d) threads\n", dimGrid.x, dimGrid.y, dimBlock.x, dimBlock.y);
	SyncKernel<<<dimGrid, dimBlock>>>();
	
	//cudaDeviceSynchronize();

	return 0;
}
