#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>

__global__ void GrayKernel( uint8_t* dev_Color_Img, uint8_t* dev_Gray_Img, int width, int height) {
    int gy = blockIdx.y * blockDim.y + threadIdx.y;
    int gx = blockIdx.x * blockDim.x + threadIdx.x;

    if (gx < width && gy < height) {
        uint8_t Luminance = 0;

        int B_idx = gx + gy * width;
        int G_idx = gx + gy * width + width * height;
        int R_idx = gx + gy * width + width * height * 2;
        
        Luminance = dev_Color_Img[R_idx] * 0.21 + dev_Color_Img[G_idx] * 0.72 + dev_Color_Img[B_idx] * 0.07;

        if (Luminance > 255) Luminance = 255;
        
        int Gray_idx = gx + gy * width;
        dev_Gray_Img[Gray_idx] = Luminance;
    }
}

void Color2Gray(uint8_t* Color_Img, uint8_t* Gray_Img, int width, int height) {
	
    uint8_t* dev_Color_Img = NULL;
    uint8_t* dev_Gray_Img = NULL;
  
    hipMalloc( (void**)&dev_Color_Img, width * height * 3 * sizeof(uint8_t) );
    hipMalloc( (void**)&dev_Gray_Img, width * height * sizeof(uint8_t) );
    
    hipMemcpy( dev_Color_Img, Color_Img, width * height * 3 * sizeof(uint8_t), hipMemcpyHostToDevice );

    dim3 dimBlock(32, 32, 1);
    dim3 dimGrid((width + dimBlock.x - 1)/ dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y, 1);
	
	clock_t start = clock();
    GrayKernel<<<dimGrid, dimBlock>>>(dev_Color_Img, dev_Gray_Img, width, height);
	clock_t end = clock();
	hipDeviceSynchronize();

    hipMemcpy( Gray_Img, dev_Gray_Img, width * height * sizeof(uint8_t), hipMemcpyDeviceToHost );
	
    hipFree( dev_Color_Img );
	hipFree( dev_Gray_Img );
	
	double execution_time = (double) (end - start) / CLOCKS_PER_SEC;
	printf("Execution time: %d usec\n", (int) (execution_time * 1000000));
}
