#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdint.h>
#include <time.h>

int width = 224;
int height = 224;
int channel = 3;



__global__ void GrayKernel( uint8_t* Gray_Img, uint8_t* Color_Img, int width, int height, int channel) {
    int gy = blockIdx.y * blockDim.y + threadIdx.y;
    int gx = blockIdx.x * blockDim.x + threadIdx.x;

    if (gx < width && gy < height) {
        uint8_t Luminance = 0;

        int B_idx = gx + gy * height;
        int G_idx = gx + gy * height + width * height;
        int R_idx = gx + gy * height + width * height * 2;
        
        Luminance = Color_Img[R_idx] * 0.21 + Color_Img[G_idx] * 0.72 + Color_Img[B_idx] * 0.07;
        if (Luminance > 255) Luminance = 255;
        
        int G_idx = gx + gy * height;
        Gray_Img[G_idx] = Luminance;
    }
}

int main(void) {
    uint8_t Color_Img[width * height];
    uint8_t Gray_Img[width * height * channel];
    
    uint8_t* dev_Color_Img = NULL;
    uint8_t* dev_Gray_Img = NULL;

    // Step 1.  
    hipMalloc( (void**)&dev_Color_Img, width * height * channel * sizeof(uint8_t) );
    hipMalloc( (void**)&dev_Gray_Img, width * height * sizeof(uint8_t) );
    
    hipMemcpy( dev_Color_Img, Color_Img, width * height * channel * sizeof(uint8_t), hipMemcpyHostToDevice );

    dim3 dimBlock(32, 32, 1);
    dim3 dimGrid((width + dimBlock.x - 1)/ dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y, 1);

    clock_t start = clock();
    GrayKernel<<<>>>;
    hipDeviceSynchronize();
    clock_t end = clock();

    hipMemcpy( dev_Gray_Img, Gray_Img, width * height * channel * sizeof(uint8_t), hipMemcpyDeviceToHost );
	
    hipFree( dev_Color_Img );
	hipFree( dev_Gray_Img );
	
    double execution_time = (double)(end - start) / CLOCKS_PER_SEC;
    
	return 0;

}
