#include "hip/hip_runtime.h"
#include "utils.c"


int matsize = 224;
int in_channel = 3;
int out_channel = 1;
int kernelsize = 3;
int padding = 0;


__global__ void kernel_gemmconv() {

}


int main(void) {
    float matA[matsize * matsize * in_channel];
    float matF[kernelsize * kernelsize * in_channel]; // Filter
    float matC[10]; // tmp
	srand( 0 );
	setNormalizedRandomData( matA, matsize * matsize * in_channel );
	setNormalizedRandomData( matF, kernelsize * kernelsize * in_channel );
    float new_matA = im2col();
    float new_matC = im2col();

    float* dev_matA = NULL;
    float* dev_matF = NULL;
    float* dev_matC = NULL;
    hipMalloc( (void**)&dev_matA, matsize * matsize * in_channel * sizeof(float) );
    hipMalloc( (void**)&dev_matF, kernelsize * kernelsize * in_channel * sizeof(float) );
    hipMalloc( (void**)&dev_matA, (matsize - (kernelsize - 1) + (2 * padding)) * (matsize - (kernelsize - 1) + (2 * padding)) * out_channel * sizeof(float) );
    hipMemcpy( dev_matA, matA, matsize * matsize * in_channel * sizeof(float), hipMemcpyHostToDevice );
    hipMemcpy( dev_matF, matF, kernelsize * kernelsize * in_channel * sizeof(float), hipMemcpyHostToDevice );


    clock_t start = clock();
    
    hipDeviceSynchronize();
    clock_t end = clock();
    hipMemcpy( dev_matC, matC, (matsize - (kernelsize - 1) + (2 * padding)) * (matsize - (kernelsize - 1) + (2 * padding)) * out_channel * sizeof(float), hipMemcpyDeviceToHost );
	hipFree( dev_matA );
	hipFree( dev_matF );
	hipFree( dev_matC );
    double execution_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Execution time: %d usec\n", (int) (execution_time * 1000000));
	printf("matrix size = nrow * ncol = %d * %d\n", matsize, matsize);
	printMat( "Feature Map", matC, matsize, matsize, out_channel );
	printMat( "matA", matA, matsize, matsize, in_channel );
	printMat( "matF", matF, matsize, matsize, in_channel );
	return 0;


}