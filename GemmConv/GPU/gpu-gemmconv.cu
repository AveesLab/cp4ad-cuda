#include "hip/hip_runtime.h"
#include "utils.cuh"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <time.h>


// 각 스레드들은 컨볼루션을 할 애들을 배열에 적어줌
__global__ void im2col_kernel( uint8_t* dev_input, uint8_t* dev_col, int col_size, int width_in, int height_in, int width_out, int height_out, int in_channel, int out_channel, int padding, int stride, int kernel_size ) {
    int gx = blockIdx.x * blockDim.x + threadIdx.x; // 0 ~ col_size

    if (gx < width_out * height_out * out_channel) {
        int start_idx = gx * kernel_size * kernel_size * in_channel;
        int cx = gx % width_out;
        int cy = gx / width_out;
        for (int in_c = 0; in_c < in_channel; in_c++) {
            for (int dy = 0; dy < kernel_size; dy++) {
                for (int dx = 0; dx < kernel_size; dx++) {
                    dev_col[start_idx] = dev_input[width_in * height_in * in_c + width_in * (dy + cy) + (dx + cx)];
                    //printf("%d = %u\n", start_idx, dev_col[start_idx]);
                    start_idx++;
                }
            }
        }
		//printf("%d\n", start_idx);
    }
}


__global__ void GEMM_kernel( uint8_t* C, uint8_t* A, int8_t* B, int width_in, int width_out, int height_out, int output_size, int kernel_size, int in_channel ) {
	// Write your code below.
	int gx = blockIdx.x * blockDim.x + threadIdx.x; 
	if (gx < output_size) {
		int sum = 0;
        int start_idx = gx * 9;
		for (int k = 0; k < 9; ++k) {
			sum += A[start_idx+k] * B[k];
		}
		if (sum < 0) sum = 0;
		else if (sum > 255) sum = 255;
		C[gx] = sum;
	}
}


void im2col_gpu( uint8_t* input, uint8_t* col, int width_in, int height_in, int width_out, int height_out, int in_channel, int out_channel, int padding, int stride, int kernel_size ) {
    uint8_t* dev_input = NULL;
    uint8_t* dev_col = NULL;
    int col_size = kernel_size * kernel_size * in_channel * width_out * height_out * out_channel; 

    hipMalloc( (void**)&dev_input, width_in * height_in * in_channel * sizeof(uint8_t) );
    hipMalloc( (void**)&dev_col, col_size * sizeof(uint8_t) );
    hipMemcpy( dev_input, input, width_in * height_in * in_channel * sizeof(uint8_t), hipMemcpyHostToDevice );

    dim3 dimBlock(512, 1, 1); // 하나의 열로만 표현 하기 위해서 x축으로만 Thread 할당
    dim3 dimGrid((col_size + dimBlock.x -1) / dimBlock.x, 1, 1);

    im2col_kernel<<<dimGrid, dimBlock>>>(dev_input, dev_col, col_size, width_in, height_in, width_out, height_out, in_channel, out_channel, padding, stride, kernel_size);
    
    hipDeviceSynchronize();
    
    hipMemcpy( col, dev_col, col_size * sizeof(uint8_t), hipMemcpyDeviceToHost );
    hipFree( dev_input );
    hipFree( dev_col );
}   


void gemm_gpu( uint8_t* input, uint8_t* output, int width_in, int height_in, int width_out, int height_out, int in_channel, int out_channel, int padding, int stride, int kernel_size ) {
    int8_t Filter[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1}; // 정답 확인을 위한 필터
    int col_size = kernel_size * kernel_size * in_channel * width_out * height_out * out_channel; 
    int output_size = width_out * height_out * out_channel;
	uint8_t* dev_input = NULL;
    int8_t* dev_Filter = NULL;
    uint8_t* dev_output = NULL;

    // Device(GPU) 메모리 할당 후 호스트 메모리에 있는 값을  Device(GPU) 메모리로 복사
    hipMalloc( (void**)&dev_input, col_size * sizeof(uint8_t) ); // im2col 변환 된 input
    hipMalloc( (void**)&dev_Filter, 9 * sizeof(int8_t) );
    hipMalloc( (void**)&dev_output, width_out * height_out * out_channel * sizeof(uint8_t) );

    hipMemcpy( dev_input, input, col_size * sizeof(uint8_t), hipMemcpyHostToDevice );
    hipMemcpy( dev_Filter, Filter, 9 * sizeof(int8_t), hipMemcpyHostToDevice );

    dim3 dimBlock(512, 1, 1);
    dim3 dimGrid((col_size + dimBlock.x - 1)/ dimBlock.x, 1, 1);
    

    clock_t start = clock();
    GEMM_kernel<<<dimGrid, dimBlock>>>(dev_output, dev_input, dev_Filter, width_in, width_out, height_out, output_size, kernel_size, in_channel );
    hipDeviceSynchronize();
    clock_t end = clock();
    hipMemcpy( output, dev_output, width_out * height_out * out_channel * sizeof(uint8_t), hipMemcpyDeviceToHost );

	hipFree( dev_input );
	hipFree( dev_Filter );
	hipFree( dev_output );
    double execution_time = (double)(end - start) / CLOCKS_PER_SEC;
    printf("Execution time: %d usec\n", (int) (execution_time * 1000000));
}
